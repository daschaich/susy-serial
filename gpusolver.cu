#include "gpusolver.h"
#include <cusp/krylov/cg_m.h>
#include <cusp/krylov/cg.h>
#include <cusp/multiply.h>
#include <cusp/print.h>
#include "utilities.h"
#include "n4st_operator.inl"

void gpusolver(Complex *m, int *col, int *row, Complex *bn, double *shift,
    Complex soln[DEGREE][LEN])
{

  // Declare cusp containers on host
<<<<<<< HEAD
  cusp::csr_matrix<int,cusp::complex<double>,cusp::host_memory> A_h(LEN,LEN,LEN*NONZEROES);
  cusp::csr_matrix<int,cusp::complex<double>,cusp::host_memory> A_h_Prime(LEN,LEN,LEN*NONZEROES);
=======
  cusp::csr_matrix<int,cusp::complex<double>,cusp::host_memory> A_h(LEN,LEN,TOTALNONZEROES);
  cusp::csr_matrix<int,cusp::complex<double>,cusp::host_memory> A_h_Prime(LEN,LEN,TOTALNONZEROES);
>>>>>>> f33135b5861f274b44c622ee0ce6ebc81e898eb0
  cusp::array1d<cusp::complex<double>, cusp::host_memory> b_h(LEN,0.0);
  cusp::array1d<double, cusp::host_memory> sigma_h(DEGREE);

  // Copy data into cusp containers
  for (int i=0; i<DEGREE; i++)
    sigma_h[i] = shift[i];
  
  for (int i = 0; i<LEN; i++)
  {
    b_h[i] = cusp::complex<double>(bn[i].real(),bn[i].imag());
    A_h.row_offsets[i] = row[i];
    A_h_Prime.row_offsets[i] = row[i];
    if(row[i] == -1){cout << "-1 detected, exiting in row" << endl; exit(1);}
  }
<<<<<<< HEAD
  for (int i=0; i<LEN*NONZEROES; i++)
=======
  for (int i=0; i<TOTALNONZEROES; i++)
>>>>>>> f33135b5861f274b44c622ee0ce6ebc81e898eb0
  {
    A_h.column_indices[i] = col[i];
    A_h_Prime.column_indices[i] = col[i];
    A_h.values[i] = cusp::complex<double>(m[i].real(),m[i].imag());
    A_h_Prime.values[i] = cusp::complex<double>(-m[i].real(),m[i].imag());
    if(col[i] == -1){cout << "-1 detected, exiting in col" << endl; exit(1);}
  }


<<<<<<< HEAD
  A_h.row_offsets[LEN] = LEN*NONZEROES;
  A_h_Prime.row_offsets[LEN] = LEN*NONZEROES;
=======
  A_h.row_offsets[LEN] = TOTALNONZEROES;
  A_h_Prime.row_offsets[LEN] = TOTALNONZEROES;
>>>>>>> f33135b5861f274b44c622ee0ce6ebc81e898eb0
  
  // Copy information to device
  cusp::csr_matrix<int,cusp::complex<double>,cusp::device_memory> A_d = A_h;
  cusp::csr_matrix<int,cusp::complex<double>,cusp::device_memory> A_d_Prime = A_h_Prime;
  cusp::array1d<cusp::complex<double>, cusp::device_memory> b_d = b_h;
  cusp::array1d<double, cusp::device_memory> sigma_d = sigma_h;
  
  // Symmetrize
  dirac_operator AA(A_d, A_d_Prime);

  // Declare the solution on the device
  cusp::array1d<cusp::complex<double>, cusp::device_memory> x_d(LEN*DEGREE, 1);

  // monitor the cg iteration silently
  cusp::default_monitor<cusp::complex<double> > monitor(b_d, 1000, 1e-14);
  
  // Solve
  cusp::krylov::cg_m(AA,x_d,b_d,sigma_d,monitor);
  
  cusp::array1d<cusp::complex<double>, cusp::host_memory> x_h = x_d;
 
  for (int n=0; n<DEGREE; n++)
  {
    for (int i=0; i<LEN; i++)
    {
      soln[n][i] = Complex(x_h[i+LEN*n].real(),x_h[i+LEN*n].imag());
    }
  }
}

